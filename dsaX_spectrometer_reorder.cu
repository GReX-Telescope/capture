#include "hip/hip_runtime.h"
// -*- c++ -*-mp1*2**cand1_widths[i]
#include <iostream>
#include <algorithm>
using std::cout;
using std::cerr;
using std::endl;
#include <stdio.h>
#include <stdlib.h>
#include <cmath>
#include <string.h>
#include <unistd.h>
#include <netdb.h>
#include <sys/socket.h>
#include <sys/types.h>
#include <netinet/in.h>
#include <time.h>

#include "dada_cuda.h"
#include "dada_client.h"
#include "dada_def.h"
#include "dada_hdu.h"
#include "multilog.h"
#include "ipcio.h"
#include "ipcbuf.h"
#include "dada_affinity.h"
#include "ascii_header.h"

#include <thrust/fill.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/functional.h>
#include <thrust/transform.h>

#include "fitsio.h"

#define NCHAN 2048
#define NANT 1

// these are in bytes
#define BLOCKSIZE 409600000
#define GULP 51200000
#define NSAMPS_GULP 12500
#define NGULPS 8

// binning for single-point flagging
#define bf 16
#define bt 25

void dsaX_dbgpu_cleanup (dada_hdu_t * hdu_in, dada_hdu_t * hdu_out, multilog_t * log);
int dada_bind_thread_to_core (int core);

// functor to do the scaling
__device__ float *s1, *s2, *s3;

struct da_functor
{

  int nant;
  da_functor(int _nant) : nant(_nant) {}

  __device__
  int operator()(const int x, const int y) const {

    int i = (int)(y % (2048)); // CHANGE FOR NANT
    //printf("x = %f",x);
    /*std::cout << "s1 = " << s1[i] <<std::endl;
    std::cout << "s2 = " << s2[i] <<std::endl;
    std::cout << "s3 = " << s3[i] <<std::endl;
    std::cout << "x s1/s2 + s3 = " << x*s1[i]/s2[i]+s3[i] <<std::endl;
    std::cout << "x s1/s2 + s3 r = " << __float2int_rn(x*s1[i]/s2[i]+s3[i]) <<std::endl;*/
    return __float2int_rn(x*s1[i]/s2[i]+s3[i]);
    
  }
};

    
int dada_bind_thread_to_core(int core)
{

  cpu_set_t set;
  pid_t tpid;

  CPU_ZERO(&set);
    CPU_SET(core, &set);
      tpid = syscall(SYS_gettid);

  if (sched_setaffinity(tpid, sizeof(cpu_set_t), &set) < 0) {
      fprintf(stderr, "failed to set cpu affinity: %s", strerror(errno));
          return -1;
	    }

  CPU_ZERO(&set);
    if ( sched_getaffinity(tpid, sizeof(cpu_set_t), &set) < 0 ) {
        fprintf(stderr, "failed to get cpu affinity: %s", strerror(errno));
	    return -1;
	      }

  return 0;
}

void usage()
{
  fprintf (stdout,
	   "dsaX_spectrometer_reorder [options]\n"
	   " -c core   bind process to CPU core\n"
	   " -d        dump 1024 spectra from each input block\n"
	   " -h        print usage\n");
}

int main (int argc, char *argv[]) {

  hipSetDevice(1);
  
  /* DADA Header plus Data Unit */
  dada_hdu_t* hdu_in = 0;
  dada_hdu_t* hdu_out = 0;

  /* DADA Logger */
  multilog_t* log = 0;

  int core = -1;

  // input data block HDU key
  key_t in_key = 0x0000dada;

  // output data block HDU key
  key_t out_key = 0x0000eada;

  // online update of BF and BT
  int BF = bf;
  int BT = bt;
  int tf, tt;
  FILE *fFT;
  
  int arg = 0;
  int dump = 0;
  int snapchoice = -1;

  while ((arg=getopt(argc,argv,"c:s:dh")) != -1)
    {
      switch (arg)
	{
	case 'c':
	  if (optarg)
	    {
	      core = atoi(optarg);
	      break;
	    }
	  else
	    {
	      fprintf (stderr, "ERROR: -c flag requires argument\n");
	      return EXIT_FAILURE;
	    }

	case 's':
	  if (optarg)
	    {
	      snapchoice = atoi(optarg);
	      break;
	    }
	  else
	    {
	      fprintf (stderr, "ERROR: -s flag requires argument\n");
	      return EXIT_FAILURE;
	    }

	case 'd':
	  dump=1;
	case 'h':
	  usage();
	  return EXIT_SUCCESS;
	}
    }
  
  // DADA stuff

  log = multilog_open ("dsaX_spectrometer_reorder", 0);

  multilog_add (log, stderr);

  multilog (log, LOG_INFO, "dsaX_spectrometer_reorder: creating in hdu\n");

  // open connection to the in/read DB
  hdu_in  = dada_hdu_create (log);
  dada_hdu_set_key (hdu_in, in_key);
  if (dada_hdu_connect (hdu_in) < 0) {
    fprintf (stderr, "dsaX_spectrometer_reorder: could not connect to dada buffer\n");
    return EXIT_FAILURE;
  }
  if (dada_hdu_lock_read (hdu_in) < 0) {
    fprintf (stderr, "dsaX_spectrometer_reorder: could not lock to dada buffer\n");
    return EXIT_FAILURE;
  }

  // open connection to the out/write DB
  hdu_out = dada_hdu_create (log);
  dada_hdu_set_key (hdu_out, out_key);
  if (dada_hdu_connect (hdu_out) < 0)
    {
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      fprintf (stderr, "dsaX_spectrometer_reorder: could not connect to eada buffer\n");
      return EXIT_FAILURE;
    }
  if (dada_hdu_lock_write(hdu_out) < 0)
    {
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      fprintf (stderr, "dsaX_spectrometer_reorder: could not lock4 to eada buffer\n");
      return EXIT_FAILURE;
    }

  if (core >= 0)
    {
      fprintf(stderr, "binding to core %d\n", core);
      if (dada_bind_thread_to_core(core) < 0)
	fprintf(stderr, "dsaX_spectrometer_reorder: failed to bind to core %d\n", core);
    }

  bool observation_complete=0;

  // more DADA stuff
  
  uint64_t header_size = 0;

  // read the header from the input HDU
  char * header_in = ipcbuf_get_next_read (hdu_in->header_block, &header_size);
  if (!header_in)
    {
      multilog(log ,LOG_ERR, "main: could not read next header\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // now write the output DADA header
  char * header_out = ipcbuf_get_next_write (hdu_out->header_block);
  if (!header_out)
    {
      multilog(log, LOG_ERR, "could not get next header block [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // read the number of stations from the header
  int nant;
  if (ascii_header_get (header_in, "NANT", "%d", &nant) != 1)
    {
      nant = NANT;
      multilog(log, LOG_WARNING, "Header with no NANT. Setting to %d\n", NANT);
    }

  fprintf(stderr, "dsaX_spectrometer_reorder: Have NANT %d\n", nant);
  fprintf(stderr, "dsaX_spectrometer_reorder: Have snapchoice %d\n", snapchoice);
  
  
  // copy the in header to the out header
  memcpy (header_out, header_in, header_size);

  // need to change some DADA parameters
  if (ascii_header_set (header_out, "NANT", "%d", 1) < 0)
    multilog(log, LOG_WARNING, "failed to set NANT 1 in header_out\n");

  // mark the input header as cleared
  if (ipcbuf_mark_cleared (hdu_in->header_block) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block cleared [input]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  // mark the output header buffer as filled
  if (ipcbuf_mark_filled (hdu_out->header_block, header_size) < 0)
    {
      multilog (log, LOG_ERR, "could not mark header block filled [output]\n");
      dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
      return EXIT_FAILURE;
    }

  //dada_cuda_dbregister (hdu_in);

  // to scale data
  thrust::host_vector<float> mult(2048*nant), sc(nant*2048), mlt(2048*nant);
  thrust::host_vector<float> mmult(2048*nant), msc(nant*2048), mmlt(2048*nant);
  thrust::device_vector<float> d_mult(2048*nant), d_sc(nant*2048), d_mlt(2048*nant);
  float *s_mult, *s_mlt, *s_sc;
  s_mult = thrust::raw_pointer_cast(&d_mult[0]);
  s_mlt = thrust::raw_pointer_cast(&d_mlt[0]);
  s_sc = thrust::raw_pointer_cast(&d_sc[0]);
  float bpscl = NSAMPS_GULP*64;

  for (int i=0;i<2048;i++) {
    for (int j=0;j<nant;j++) {
      sc[i+j*2048] = 0.;
      mult[i+j*2048] = bpscl;
      if (i<330) mult[i+j*2048]=0.;
//if (i>1354 && i<1367) mult[i+j*2048] = 0.;
//if (i>1982 && i<1986) mult[i+j*2048] = 0.;
//if (i>1965 && i<1974) mult[i+j*2048] = 0.;
//if (i>1920 && i<1935) mult[i+j*2048] = 0.;
/*if (i>1748 && i<1764) mult[i+j*2048] = 0.;
if (i>1704 && i<1724) mult[i+j*2048] = 0.; 
if (i>1673 && i<1691) mult[i+j*2048] = 0.;
if (i>1629 && i<1651) mult[i+j*2048] = 0.;
if (i>1359 && i<1593) mult[i+j*2048] = 0.;
if (i>1056 && i<1243) mult[i+j*2048] = 0.;
if (i>988 && i<1028) mult[i+j*2048] = 0.;
if (i>541 && i<543) mult[i+j*2048] = 0.;
    if (i>2033) mult[i+j*2048]=0.;*/
/*if (i>1354 && i<1367) mult[i+j*2048] = 0.;      
if (i>1528 && i<1547) mult[i+j*2048] = 0.;
if (i>1570 && i<1589) mult[i+j*2048] = 0.;
if (i>1207 && i<1220) mult[i+j*2048] = 0.;
if (i>515 && i<552) mult[i+j*2048] = 0.;
      if (i>1208 && i<1232) mult[i+j*2048] = 0.;
      if (i>1466 && i<1482) mult[i+j*2048] = 0.;
      if (i>1777 && i<1794) mult[i+j*2048] = 0.;*/
      //if (i>1888) mult[i+j*2048] = 0.;
      if (mult[i+j*2048]==0.) mlt[i+j*2048]=64.;
      else mlt[i+j*2048] = 0.;
    }
  }
   
  
  /*hipMemcpyToSymbol(HIP_SYMBOL(s1),&s_mult,sizeof(float *));
  hipMemcpyToSymbol(HIP_SYMBOL(s2),&s_sc,sizeof(float *));
  hipMemcpyToSymbol(HIP_SYMBOL(s3),&s_mlt,sizeof(float *));*/

  // file for logging flagged spectra
  fitsfile *fptr;
  char fitsnam[100];
  int status=0;
  int rownum = 1;
  time_t rawtime;
  struct tm *info;
  time(&rawtime);
  info = localtime(&rawtime);
  //double MJD = (double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
  //char MJD[100];
  long double MJD;
   if (ascii_header_get (header_in, "MJD_START", "%Lf", &MJD) != 1)
    {
      MJD = (long double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
      //MJD = (char*)(&MJD_double);
      multilog(log, LOG_WARNING, "Header with no MJD_START. Setting to %s\n", &MJD);
    }
float MJD_f = (float)MJD;
//printf("reorder: %0.30Lf\n",MJD);
 
  sprintf(fitsnam,"/home/user/data/slog_%0.3Lf.fits",MJD);
  char *ttype[] = {"Spectra","Perc_ts","Perc_samp"};
  char *tform[] = {"2048E", "E", "E"}; // EDIT FOR NANT
  char *tunit[] = {"\0", "\0", "\0"};
  char extname[] = "spec_log";
  fits_create_file(&fptr, fitsnam, &status);
  if (status) cerr << "create_file FITS error " << status << endl;
  fits_create_tbl(fptr, BINARY_TBL, 0, 3, ttype, tform, tunit, extname, &status);
  if (status) cerr << "create_tbl FITS error " << status << endl;
  fits_write_key(fptr, TFLOAT, "MJD", &MJD_f, "Start MJD", &status);
  float mytsamp = NSAMPS_GULP*6.5536e-5*2;
  fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);

  if (status) cerr << "FITS error " << status << endl;
  else
    cout << "Opened FITS file " << fitsnam << endl;
  fits_close_file(fptr, &status);
  float out_bp[nant*2048], out_pts[1], out_psamp[1];

  // setu up vars
  uint64_t block_size = ipcbuf_get_bufsz ((ipcbuf_t *) hdu_in->data_block);
  uint64_t bytes_to_read;
  uint64_t block_id;
  char *   block;
  int bytes_to_write = BLOCKSIZE;
  uint64_t bytes_written=0;
  uint64_t written=0;
  int ibyte, idx, idxo, idxs, idxg;
  
  // allocate memory to output
  unsigned short * out_data;
  thrust::host_vector<int> h_gulpdata(2048*nant*NSAMPS_GULP);
  thrust::device_vector<int> d_gulpdata(2048*nant*NSAMPS_GULP);
  thrust::device_vector<int> d_idx(2048*nant*NSAMPS_GULP);
  thrust::sequence(d_idx.begin(),d_idx.end());
  out_data = (unsigned short *)malloc(sizeof(unsigned short)*BLOCKSIZE/2);
  char *h_indata;
  uint64_t  bytes_read = 0, clipped, clipped_ts;
  int bp[2048*nant], old_bp[2048*nant], ts[NSAMPS_GULP];
  int ts_sum, ds_sum;
  int thresh_ts = (int)((2048*nant*64*BT)+(4.*sqrt(2048.*nant*BT)*10.));
  float thresh_diff = 10*(0.32/sqrt(NSAMPS_GULP*1.));
  //unsigned short repval = (unsigned short)(av_bp);
  //if (snapchoice!=-1)
  //  repval = (unsigned short)(av_bp);
  int clipthresh = (int)(64.*nant*BT*BF+50000.*sqrt(1.*nant*BT*BF))*100000;
  int clipthresh_ss = (int)(64.*nant+45.*sqrt(1.*nant));
  if (snapchoice!=-1)
    clipthresh = 104000000*200*200000;
  unsigned short tmp;
  int intct = 0;
  int started_recording = 0;
  char cmd[200];
  uint64_t specnum = 0;
  int nints=0;
  
  multilog(log, LOG_INFO, "main: starting observation\n");
  
  while (!observation_complete) {

    if (nints > 4395) {

      rownum = 1;
      nints=0;
      time(&rawtime);
      info = localtime(&rawtime);
      MJD = (double)(57754.+info->tm_yday+(info->tm_hour+8.)/24.+info->tm_min/(24.*60.)+info->tm_sec/(24.*60.*60.));
      sprintf(fitsnam,"/home/user/data/slog_%.3Lf.fits",MJD);
      char *ttype[] = {"Spectra","Perc_ts","Perc_samp"};
      char *tform[] = {"2048E", "E", "E"}; // EDIT FOR NANT
      char *tunit[] = {"\0", "\0", "\0"};
      char extname[] = "spec_log";
      fits_create_file(&fptr, fitsnam, &status);
      if (status) cerr << "create_file FITS error " << status << endl;
      fits_create_tbl(fptr, BINARY_TBL, 0, 3, ttype, tform, tunit, extname, &status);
      if (status) cerr << "create_tbl FITS error " << status << endl;
      fits_write_key(fptr, TFLOAT, "MJD", &MJD_f, "Start MJD", &status);
      fits_write_key(fptr, TFLOAT, "TSAMP", &mytsamp, "Sample time (s)", &status);

      if (status) cerr << "FITS error " << status << endl;
      else
	cout << "Opened FITS file " << fitsnam << endl;
      fits_close_file(fptr, &status);

    }

    // deal with BF and BT
    if ((fFT=fopen("/home/user/runtime/fFT.dat","r"))) {
      fscanf(fFT,"%d %d\n",&tf,&tt);
      if (tf!=BF) {
	multilog(log, LOG_INFO, "main: new BF %d\n", tf);
	BF=tf;
      }
      if (tt!=BT) {
	multilog(log, LOG_INFO, "main: new BT %d\n", tt);
	BT=tt;
      }
      fclose(fFT);
    } 
    for (int gulp=0;gulp<NGULPS;gulp++) {
//cout << "gulp = " << gulp << std::endl;
      // read a DADA block
      h_indata = ipcio_open_block_read (hdu_in->data_block, &bytes_read, &block_id);
//cout << "h_indata size" << strlen(h_indata) << std::endl;
      //multilog(log, LOG_INFO, "main: read block which contains %lld bytes\n", bytes_read);
      
      /* Assuming mean of 64, each 8x20-sample point will have stddev of 10. 
	 Then, over NSAMPS samples, the stddev is 10/sqrt(NSAMPS*nant). Can 
	 flag every 10-sigma channel, and every 6-sigma burst */

      // deal with zero-ing stuff and setting up bandpasses.
      thrust::fill(h_gulpdata.begin(),h_gulpdata.end(),0);
      clipped = 0;
      clipped_ts = 0;
      for (int i=0;i<NSAMPS_GULP;i++) ts[i] = 0;
      for (int i=0;i<2048*nant;i++) {
	if (started_recording) old_bp[i]=bp[i];
	else old_bp[i] = 0;
	bp[i] = 0;
      }
    int sum_bp = 0;
    int av_bp_n = 0;
    int av_bp;
      // unpack data into h_gulpdata, and find current bp
      for (int k=0;k<NSAMPS_GULP;k++) {
	
	for (int snp=0;snp<nant;snp++) {
	  
	  for (int i=0;i<512;i++) {
	    for (int j=0;j<4;j++) {
	    
	      idx = k*nant*4096+snp*4096+i*8+j*2;
	      idxg = k*nant*2048+snp*2048+i*4+j;
	      idxs = i*4+j+snp*2048;
	      tmp=0;
	      tmp |= (unsigned short)(h_indata[idx]) << 8;
	      //cout << "tmp1 " << tmp << std::endl;
	      tmp |= (unsigned short)(h_indata[idx+1]);
	      //cout << "tmp2 " << tmp << std::endl;
	      //cout << "h_indata[idx] = " << (unsigned short)h_indata[idx] << std::endl;
              //cout << "h_indata[idx+1] = " << (unsigned short)h_indata[idx+1] << std::endl;
	      h_gulpdata[idxg] = (int)tmp;
	      //cout << "h_gulpdata[idxg] = " << (int)tmp << std::endl;
	      bp[idxs] += (int)tmp;
	      sum_bp += (int)tmp;
	      av_bp_n+=1;
//if(i==247 && k == 1) cout << int(tmp) << std::endl;
	    }
	    
	  }
	
	}
	
      }

av_bp = sum_bp/av_bp_n;
unsigned short repval = 64.;
int chans_clipped = 0;
int bp_clipped = 0;
int spec_clipped = 0;
//cout << "av_bp " << av_bp <<std::endl;
      // set up scaling by bandpass
      for (int i=0;i<2048*nant;i++) {
	mmult[i] = mult[i];
	mmlt[i] = mlt[i];
	msc[i] = bp[i]*1.;
	if (old_bp[i]==0) {
	  if (chans_clipped < 820) {
	  	mmult[i] = 0.;
	  	mmlt[i] = 64.;
	  	bp_clipped += 1;
	  	chans_clipped +=1;
	  }
	}
	else if ((((float)(bp[i]-old_bp[i]))/((float)(old_bp[i]))>thresh_diff) || (((float)(old_bp[i]-bp[i]))/((float)(old_bp[i]))>thresh_diff)) {
		if (chans_clipped < 820) {
			//mult[i] = 0;
			//mlt[i] = 64.;
	  		mmult[i] = 0.;
	  		mmlt[i] = 64.;
	  		chans_clipped += 1.;
			bp_clipped += 1.;
		}
	}
      }
      thrust::copy(mmult.begin(),mmult.end(),d_mult.begin());
      thrust::copy(mmlt.begin(),mmlt.end(),d_mlt.begin());
      thrust::copy(msc.begin(),msc.end(),d_sc.begin());
      s_mult = thrust::raw_pointer_cast(&d_mult[0]);
      s_mlt = thrust::raw_pointer_cast(&d_mlt[0]);
      s_sc = thrust::raw_pointer_cast(&d_sc[0]);
      hipMemcpyToSymbol(HIP_SYMBOL(s1),&s_mult,sizeof(float *));
      hipMemcpyToSymbol(HIP_SYMBOL(s2),&s_sc,sizeof(float *));
      hipMemcpyToSymbol(HIP_SYMBOL(s3),&s_mlt,sizeof(float *));
	//std::cout << "s1 = " << mmult[1000] << std::endl;
	//std::cout << "s2 = " << msc[1000] << std::endl;
	//std::cout << "s3 = " << mmlt[1000] << std::endl;
	//std::cout << "x = " << h_gulpdata[1000] << std::endl;
	//std::cout << "x*s1/s2+s3 = " << h_gulpdata[1000]*mmult[1000]/msc[1000]+mlt[1000] << std::endl;
      // do bandpass scaling of data
      thrust::copy(h_gulpdata.begin(),h_gulpdata.end(),d_gulpdata.begin());
      thrust::transform(d_gulpdata.begin(),d_gulpdata.end(),d_idx.begin(),d_gulpdata.begin(),da_functor(nant));
      thrust::copy(d_gulpdata.begin(),d_gulpdata.end(),h_gulpdata.begin());

      // reduce to out_data, and find ts
      for (int k=0;k<NSAMPS_GULP;k++) {
	for (int snp=0;snp<nant;snp++) {    
	  for (int i=0;i<2048;i++) {
	    idxo = (k+gulp*NSAMPS_GULP)*2048+i;
	    idxg = k*nant*2048+snp*2048+i;
	    if (snapchoice !=-1) {
	      if (snp==snapchoice) out_data[idxo] = (unsigned short)(h_gulpdata[idxg]);
	    }
	    else {
	      if (snp==0) out_data[idxo] = (unsigned short)(h_gulpdata[idxg]);
	      else out_data[idxo] = out_data[idxo] + (unsigned short)(h_gulpdata[idxg]);
	    }
	    ts[k] += (h_gulpdata[idxg]);
	  }
	}
      }
int max_clipped = 0;
      if (snapchoice==-1) {
	for (int k=0;k<NSAMPS_GULP;k++) {
	  int ind_spec_clipped = 0; 
	  for (int i=0;i<2048;i++) {
	   if (out_data[(k+gulp*NSAMPS_GULP)*2048+i]>clipthresh_ss && ind_spec_clipped+bp_clipped < 820) {
	      out_data[(k+gulp*NSAMPS_GULP)*2048+i]=repval;
	      clipped++;
	      chans_clipped++;
	      spec_clipped++;
	      ind_spec_clipped++;
	    }
	}
		if (ind_spec_clipped > max_clipped) {
			max_clipped = ind_spec_clipped;
		}
	  
	  
	}
      }
   std::cout << "Channels clipped: " << bp_clipped+max_clipped<< std::endl;
std::cout << "bp clipped: " << bp_clipped << std::endl;
std::cout << "Spec clipped: " << max_clipped << std::endl;

      if (snapchoice==-1) {
	for (int k=0;k<NSAMPS_GULP/BT;k++) {

	  // time-series flagging
	  ts_sum = 0;
	  /*for (int i=k*BT;i<(k+1)*BT;i++)
	    ts_sum += ts[i];
	  if (ts_sum>thresh_ts) {
	    //cout << ts_sum << " " << thresh_ts << endl;
	    clipped_ts+=BT;
	    for (int j=k*BT;j<(k+1)*BT;j++) {
	      for (int i=0;i<2048;i++) 
		out_data[(j+gulp*NSAMPS_GULP)*2048+i] = repval;
	    }
	  }*/
	  // single-point flagging
	  for (int i=0;i<2048/BF;i++) {
	    ds_sum = 0;
	    for (int j=k*BT;j<(k+1)*BT;j++) {
	      for (int l=i*BF;l<(i+1)*BF;l++)
		ds_sum += out_data[(j+gulp*NSAMPS_GULP)*2048+l];
	    }
	    if (ds_sum>clipthresh) {
	      for (int j=k*BT;j<(k+1)*BT;j++) {
		//for (int l=i*BF;l<(i+1)*BF;l++)
		  //out_data[(j+gulp*NSAMPS_GULP)*2048+l] = repval;
		//clipped+=BT*BF;
	      }
	    }
	  }
	 if(k==0) {
                for(int i=0;i<2048;i++) {
                        out_data[(gulp*NSAMPS_GULP)*2048+i] = repval + 0.3;
                }
          } 
	}
      }
      
    
      // do logging
      //multilog(log, LOG_INFO, "main: processed %d bytes, clipped percentage %.10f, samples %lld, with zero-DM clipping %lld of %d\n", GULP, (float)(100.*clipped/(GULP/2.)), clipped, clipped_ts, NSAMPS_GULP);
//cout << "clipped percentage " << (float)(100.*clipped/(NSAMPS_GULP*2048)) << std::endl;
      fits_open_table(&fptr, fitsnam, READWRITE, &status);
      for (int i=0;i<2048*nant;i++) {
	//if (mmult[i]!=0) out_bp[i] = 1.*bp[i];
	if (bp[i]>=0) out_bp[i] = 1.*bp[i];  
	else out_bp[i] = -1.*bp[i];
      }
      out_pts[0] = (float)(100.*clipped_ts/(NSAMPS_GULP));
      out_psamp[0] = (float)(100.*clipped/(NSAMPS_GULP*2048));
      fits_write_col(fptr, TFLOAT, 1, rownum, 1, nant*2048, out_bp, &status);
      fits_write_col(fptr, TFLOAT, 2, rownum, 1, 1, out_pts, &status);
      fits_write_col(fptr, TFLOAT, 3, rownum, 1, 1, out_psamp, &status);
      if (status) cerr << "FITS error in write " << status << endl;

      rownum += 1;
      fits_update_key(fptr, TINT, "NAXIS2", &rownum, "", &status);
      fits_close_file(fptr, &status);
      nints++;
      
      /*if (intct>0) flog = fopen("/mnt/nfs/data/spectrometer_log.dat","a");
      for (int i=0;i<2048*nant;i++) {
	if (mmult[i]!=0) fprintf(flog,"%d\n",(int)(old_bp[i]));
	else fprintf(flog,"%d\n",-(int)(old_bp[i]));
	//fprintf(flog,"%d\n",bp[i]-old_bp[i]);
      }
      fprintf(flog,"%g\n",(float)(100.*clipped/(GULP/2.)));
      fprintf(flog,"%g\n",(float)(100.*clipped_ts/(NSAMPS_GULP)));
      fclose(flog);*/
      intct++;

      // do the start
      started_recording = 1;

      // close block for reading
      ipcio_close_block_read (hdu_in->data_block, bytes_read);
//cout << "Done with gulp" << std::endl;
      if (bytes_read < block_size) {
	observation_complete = 1;
	multilog(log, LOG_INFO, "main: finished, with bytes_read %llu < expected %llu\n", bytes_read, block_size);
	break;
      }
      
    }
    // DO THE WRITING
    written = ipcio_write (hdu_out->data_block, (char *) out_data, bytes_to_write);
    
    if (written < bytes_to_write)
      {
	multilog(log, LOG_INFO, "main: failed to write all data to datablock [output]\n");
	dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
	return EXIT_FAILURE;
      }
    bytes_written += written;
    cout << "Wrote " << bytes_written << " bytes"  << endl;
    //specnum += 100000;
    //multilog(log, LOG_INFO, "main: write %llu bytes, %llu total, specnum %llu\n", written, bytes_written, specnum*16-1000000);
    
    //sprintf(cmd,"echo p %lu | nc -4u -w1 10.10.1.11 11223",specnum);
    //system(cmd);
    
  }
    
  dsaX_dbgpu_cleanup (hdu_in, hdu_out, log);
  free(out_data);
  
}

void dsaX_dbgpu_cleanup (dada_hdu_t * in, dada_hdu_t * out, multilog_t * log)
{

  //dada_cuda_dbunregister (in);
  
  if (dada_hdu_unlock_read (in) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock read on hdu_in\n");
    }
  dada_hdu_destroy (in);

  if (dada_hdu_unlock_write (out) < 0)
    {
      multilog(log, LOG_ERR, "could not unlock write on hdu_out\n");
    }
  dada_hdu_destroy (out);
}
